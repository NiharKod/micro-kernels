#include <iostream>
#include <hip/hip_runtime.h>
#include <microkernels/elementwise.cuh>

void check_cuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at " << msg << ": " << hipGetErrorString(err) << "\n";
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int N = 10;
    float h_in[N], h_out[N];

    // Fill input with values -5 to 4
    for (int i = 0; i < N; ++i) {
        h_in[i] = i - 5;
    }

    float *d_in = nullptr, *d_out = nullptr;
    check_cuda(hipMalloc(&d_in, N * sizeof(float)), "hipMalloc d_in");
    check_cuda(hipMalloc(&d_out, N * sizeof(float)), "hipMalloc d_out");

    check_cuda(hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy H2D");

    // Launch the ReLU kernel
    microkernels::launch_elementwise(d_out, d_in, N, microkernels::ReLU());

    // Ensure kernel launch was okay
    check_cuda(hipGetLastError(), "kernel launch");

    check_cuda(hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy D2H");

    check_cuda(hipFree(d_in), "hipFree d_in");
    check_cuda(hipFree(d_out), "hipFree d_out");

    std::cout << "Input:\n";
    for (int i = 0; i < N; ++i) std::cout << h_in[i] << " ";
    std::cout << "\n";

    std::cout << "ReLU Output:\n";
    for (int i = 0; i < N; ++i) std::cout << h_out[i] << " ";
    std::cout << "\n";

    return 0;
}
